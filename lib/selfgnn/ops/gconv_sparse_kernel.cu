#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>
#include "gconv.h"
#include "common.cuh"


template <int blockSize, typename scalar_t>
__global__ void gconv_sparse_forward_kernel(
    const scalar_t* __restrict__ feature,
    const int* __restrict__ edge_type,
    const scalar_t* __restrict__ filter,
    scalar_t* __restrict__ res,
    size_t nin_feature,
    size_t nedge_type,
    size_t elist_length)
{
    extern __shared__ int s [];
    int batch_sz = blockIdx.x;
    int node_id = blockIdx.y;
    int nout = blockIdx.z;

    int nnodes = gridDim.y;

    int tid = threadIdx.x;

    volatile scalar_t * partial_res = (scalar_t *) s;
    const scalar_t* cfeature = feature + batch_sz * (nnodes * nin_feature);
    const scalar_t* kernels = filter + nin_feature * nedge_type * nout;
    const int nneibours = edge_type[(batch_sz * nnodes + node_id) * (elist_length)];
    const int *edge_list = edge_type + (batch_sz * nnodes + node_id) * (elist_length) + 1;

    scalar_t* cres = res + batch_sz * nnodes * gridDim.z + node_id * gridDim.z + nout;

    
    partial_res[tid] = 0.0;

    if(tid >= nin_feature) return;
    for(int edge_id = 0; edge_id < nneibours; edge_id++){
        int cnode_id = edge_list[2 * edge_id];
        int edge_type_ = edge_list[2 * edge_id + 1];
        for(int cfeature_id = tid; cfeature_id < nin_feature; cfeature_id+= blockDim.x){
            partial_res[tid] += kernels[cfeature_id * nedge_type + edge_type_] * cfeature[cnode_id * nin_feature + cfeature_id];
        }
    }

    __syncthreads();
    if (blockSize >= 512) { 
        if (tid < 256) { partial_res[tid] += partial_res[tid + 256]; }
        __syncthreads(); }
    if (blockSize >= 256) { 
        if (tid < 128) { partial_res[tid] += partial_res[tid + 128]; }
        __syncthreads(); }
    if (blockSize >= 128) { 
        if (tid <  64) { partial_res[tid] += partial_res[tid + 64]; }
        __syncthreads(); }
    if (tid < 32) warpReduceSum<blockSize, scalar_t>(partial_res, tid);

    if (tid == 0){
        cres[0] = partial_res[0];
    }
}


#define run_kernel(bsize, type) gconv_sparse_forward_kernel<bsize, type><<<blocks, threads, threads * sizeof(type)>>> (feature, edge_type,  filter,  res, nin_features, nedge_type, elist_length)
    
template <typename scalar_t>
void gconv_sparse_forward_runner(const scalar_t * feature,
                                 const int * edge_type,
                                 const scalar_t * filter,
                                 scalar_t * res,
                                 size_t batch_size,
                                 size_t nnodes,
                                 size_t nin_features,
                                 size_t nout_features,
                                 size_t nedge_type,
                                 size_t elist_length)
{
    int best_num_threads = pow(2, ceil(log(nin_features)/log(2)));
    int threads = best_num_threads > 512?512:best_num_threads ;

    const dim3 blocks(batch_size, nnodes, nout_features);

    switch(threads){

    case 512:
        run_kernel(512, scalar_t);
        break;
    case 256:
        run_kernel(256, scalar_t);
        break;
    case 128:
        run_kernel(128, scalar_t);
        break;
    case 64:
        run_kernel(64, scalar_t);
        break;
    case 32:
        run_kernel(32, scalar_t);
        break;
    case 16:
        run_kernel(16, scalar_t);
        break;
    case 8:
        run_kernel(8, scalar_t);
        break;
    case 4:
        run_kernel(4, scalar_t);
        break;
    case 2:
        run_kernel(2, scalar_t);
        break;
    case 1:
        run_kernel(1, scalar_t);
        break;
    }
}

#undef run_kernel


#define scalar_t double
template 
void gconv_sparse_forward_runner(const scalar_t * feature,
                                 const int * edge_type,
                                 const scalar_t * filter,
                                 scalar_t * res,
                                 size_t batch_size,
                                 size_t nnodes,
                                 size_t nin_features,
                                 size_t nout_features,
                                 size_t nedge_type,
                                 size_t elist_length);
#undef scalar_t

#define scalar_t float
template
void gconv_sparse_forward_runner(const scalar_t * feature,
                                 const int * edge_type,
                                 const scalar_t * filter,
                                 scalar_t * res,
                                 size_t batch_size,
                                 size_t nnodes,
                                 size_t nin_features,
                                 size_t nout_features,
                                 size_t nedge_type,
                                 size_t elist_length);
#undef scalar_t


