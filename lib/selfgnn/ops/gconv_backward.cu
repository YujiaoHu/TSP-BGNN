#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>
#include "gconv.h"
#include "common.cuh"


template <int blockSize, typename scalar_t>
__global__ void gconv_cuda_backward_kernel_filter(
    const scalar_t* __restrict__ grad_res,
    const scalar_t* __restrict__ feature,
    const int* __restrict__ edge_type,
    scalar_t * __restrict__ filter_grad,
    size_t batch_sz,
    size_t nnodes
    )
{
    extern __shared__ int s [];
    int ou_id = blockIdx.x;
    int in_id = blockIdx.y;
    // 
    int filter_id = blockIdx.z;

    int nou_feature = gridDim.x;
    int nin_feature = gridDim.y;
    int nedge_type = gridDim.z;

    volatile scalar_t * partial_res = (scalar_t *) s;
    scalar_t * cres = filter_grad + (ou_id * nin_feature + in_id)* nedge_type + filter_id;

    int tid = threadIdx.x;
    partial_res[tid] = 0.0;
    
    if (tid >= nnodes * nnodes * batch_sz) return;

    for(int i = tid; i < nnodes * nnodes * batch_sz; i += blockDim.x){
        int batch_id = i / (nnodes * nnodes);
        int ou_node_id = (i % (nnodes * nnodes)) / nnodes;
        int in_node_id = i % nnodes;

        int edge_type_ = edge_type[i];
        if(edge_type_ == filter_id){
            int cin_id = batch_id * (nnodes * nin_feature) + in_node_id * nin_feature + in_id;
            int cou_id = batch_id * (nnodes * nou_feature) + ou_node_id * nou_feature + ou_id;
            partial_res[tid] += grad_res[cou_id] * feature[cin_id];
        }
    }

    __syncthreads();
    if (blockSize >= 512) { 
        if (tid < 256) { partial_res[tid] += partial_res[tid + 256]; }
        __syncthreads(); }
    if (blockSize >= 256) { 
        if (tid < 128) { partial_res[tid] += partial_res[tid + 128]; }
        __syncthreads(); }
    if (blockSize >= 128) { 
        if (tid <  64) { partial_res[tid] += partial_res[tid + 64]; }
        __syncthreads(); }
    if (tid < 32) warpReduceSum<blockSize, scalar_t>(partial_res, tid);

    if (tid == 0){
        cres[0] = partial_res[0];
    }

}



template <int blockSize, typename scalar_t>
__global__ void gconv_cuda_backward_kernel_indata(
    const scalar_t* __restrict__ grad_res,
    const scalar_t* __restrict__ filter,
    const int* __restrict__ edge_type,
    scalar_t * __restrict__ feature_grad,
    size_t nou_feature,
    size_t nedge_type
    )
{
    extern __shared__ int s [];
    int batch_id = blockIdx.x;
    int node_id = blockIdx.y;
    int nin = blockIdx.z;

    int nnodes = gridDim.y;
    int nin_feature = gridDim.z;

    volatile scalar_t * partial_res = (scalar_t *) s;

    const scalar_t* out_feature_grad = grad_res + batch_id * nnodes * nou_feature;
    const int * cedge_type = edge_type + batch_id * nnodes * nnodes;

    scalar_t * cres = feature_grad + (batch_id * nnodes + node_id) * nin_feature + nin;
    int tid = threadIdx.x;
    
    partial_res[tid] = 0.0;
    if(tid >= nou_feature * nnodes) return;
    
    for(int i = tid; i < nou_feature * nnodes; i += blockDim.x){
        int cnode_id = i / nou_feature;
        int cfeature_id = i % nou_feature;
        int edge_type_ = cedge_type[cnode_id * nnodes + node_id];
        if(edge_type_ >= 0){
            partial_res[tid] += filter[(cfeature_id * nin_feature + nin) * nedge_type + edge_type_] * out_feature_grad[i];
        }
    }

    __syncthreads();
    if (blockSize >= 512) { 
        if (tid < 256) { partial_res[tid] += partial_res[tid + 256]; }
        __syncthreads(); }
    if (blockSize >= 256) { 
        if (tid < 128) { partial_res[tid] += partial_res[tid + 128]; }
        __syncthreads(); }
    if (blockSize >= 128) { 
        if (tid <  64) { partial_res[tid] += partial_res[tid + 64]; }
        __syncthreads(); }
    if (tid < 32) warpReduceSum<blockSize, scalar_t>(partial_res, tid);

    if (tid == 0){
        cres[0] = partial_res[0];
    }
}


#define run_kernel(bsize, type) gconv_cuda_backward_kernel_filter<bsize, type><<< blocks, bsize, bsize * sizeof(type)  >>> (grad_res, feature, edge_type, filter_grad, batch_size, nnodes)

template <typename scalar_t>
void gconv_cuda_backward_filter_runner(const scalar_t * grad_res,
                                       const scalar_t *feature,
                                       const int * edge_type,
                                       scalar_t * filter_grad,
                                       size_t batch_size,
                                       size_t nnodes,
                                       size_t nin_features,
                                       size_t nout_features,
                                       size_t nedge_type
    )
{
    int best_num_threads = pow(2, ceil(log(batch_size * nnodes * nnodes)/log(2)));
    int threads = best_num_threads > 512?512:best_num_threads ;
    const dim3 blocks(nout_features, nin_features, nedge_type);
    switch(threads){

    case 512:
        run_kernel(512, scalar_t);
        break;
    case 256:
        run_kernel(256, scalar_t);
        break;
    case 128:
        run_kernel(128, scalar_t);
        break;
    case 64:
        run_kernel(64, scalar_t);
        break;
    case 32:
        run_kernel(32, scalar_t);
        break;
    case 16:
        run_kernel(16, scalar_t);
        break;
    case 8:
        run_kernel(8, scalar_t);
        break;
    case 4:
        run_kernel(4, scalar_t);
        break;
    case 2:
        run_kernel(2, scalar_t);
        break;
    case 1:
        run_kernel(1, scalar_t);
        break;
    }

}
#undef run_kernel


template 
void gconv_cuda_backward_filter_runner(const double * grad_res,
                                       const double *feature,
                                       const int * edge_type,
                                       double * filter_grad,
                                       size_t batch_size,
                                       size_t nnodes,
                                       size_t nin_features,
                                       size_t nout_features,
                                       size_t nedge_type
    );


template 
void gconv_cuda_backward_filter_runner(const float * grad_res,
                                       const float *feature,
                                       const int * edge_type,
                                       float * filter_grad,
                                       size_t batch_size,
                                       size_t nnodes,
                                       size_t nin_features,
                                       size_t nout_features,
                                       size_t nedge_type
    );



#define run_kernel(bsize, type) gconv_cuda_backward_kernel_indata<bsize, type>  <<<blocks, bsize, bsize * sizeof(scalar_t)  >>>(grad_res, filter, edge_type, feature_grad, nout_features, nedge_type)

template <typename scalar_t>
void gconv_cuda_backward_kernel_indata_runner(
    const scalar_t *grad_res,
    const scalar_t *filter,
    const int * edge_type,
    scalar_t * feature_grad,
    size_t batch_size,
    size_t nnodes,
    size_t nin_features,
    size_t nout_features,
    size_t nedge_type
    )
{
    int best_num_threads = pow(2, ceil(log(nout_features * nnodes)/ log(2)));
    int threads = best_num_threads > 512?512:best_num_threads ;
    const dim3 blocks(batch_size, nnodes, nin_features);
    switch(threads){

    case 512:
        run_kernel(512, scalar_t);
        break;
    case 256:
        run_kernel(256, scalar_t);
        break;
    case 128:
        run_kernel(128, scalar_t);
        break;
    case 64:
        run_kernel(64, scalar_t);
        break;
    case 32:
        run_kernel(32, scalar_t);
        break;
    case 16:
        run_kernel(16, scalar_t);
        break;
    case 8:
        run_kernel(8, scalar_t);
        break;
    case 4:
        run_kernel(4, scalar_t);
        break;
    case 2:
        run_kernel(2, scalar_t);
        break;
    case 1:
        run_kernel(1, scalar_t);
        break;
    }
}



template
void gconv_cuda_backward_kernel_indata_runner(
    const double *grad_res,
    const double *filter,
    const int * edge_type,
    double * feature_grad,
    size_t batch_size,
    size_t nnodes,
    size_t nin_features,
    size_t nout_features,
    size_t nedge_type
    );



template
void gconv_cuda_backward_kernel_indata_runner(
    const float *grad_res,
    const float *filter,
    const int * edge_type,
    float * feature_grad,
    size_t batch_size,
    size_t nnodes,
    size_t nin_features,
    size_t nout_features,
    size_t nedge_type
    );
